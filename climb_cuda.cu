/*
 * Grupo 17
 * Juan Antonio Pages Lopez y Sergio Sanz Sanz
 *
 *
 * Probabilistic approach to locate maximum heights
 * Hill Climbing + Montecarlo
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2021/2022
 *
 * v1.1
 *
 * (c) 2022 Arturo Gonzalez Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<math.h>
#include<limits.h>
#include<sys/time.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL( a )	{ \
	hipError_t ok = a; \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}
#define CHECK_CUDA_LAST()	{ \
	hipError_t ok = hipGetLastError(); \
	if ( ok != hipSuccess ) \
		fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString( ok ) ); \
	}


#define	PRECISION	10000

/* 
 * Structure to represent a climbing searcher 
 * 	This structure can be changed and/or optimized by the students
 */
typedef struct {
	int id;				// Searcher identifier
	int pos_row, pos_col;		// Position in the grid
	int steps;			// Steps count
	int follows;			// When it finds an explored trail, who searched that trail
} Searcher;


/* 
 * Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/* 
 * Macro function to simplify accessing with two coordinates to a flattened array
 * 	This macro-function can be changed and/or optimized by the students
 */
#define accessMat( arr, exp1, exp2 )	arr[ (int)(exp1) * columns + (int)(exp2) ]




/*
 * Function: Generate height for a given position
 * 	This function can be changed and/or optimized by the students
 */
__device__ int get_height( int x, int y, int rows, int columns, float x_min, float x_max, float y_min, float y_max  ) {
	/* Calculate the coordinates of the point in the ranges */
	float x_coord = x_min + ( (x_max - x_min) / rows ) * x;
	float y_coord = y_min + ( (y_max - y_min) / columns ) * y;
	/* Compute function value */
	float value = 2 * sin(x_coord) * cos(y_coord/2) + log( fabs(y_coord - M_PI_2) );
	/* Transform to fixed point precision */
	int fixed_point = (int)( PRECISION * value );
	return fixed_point;
}

/*
 * Function: Climbing step
 * 	This function can be changed and/or optimized by the students
 */
__device__ int climbing_step( int rows, int columns, Searcher *searchers, int search, int *heights, int *trails, int *tainted, float x_min, float x_max, float y_min, float y_max ) {
	int search_flag = 0;

	/* Annotate one step more, landing counts as the first step */
	searchers[ search ].steps ++;

	/* Get starting position */
	int pos_row = searchers[ search ].pos_row;
	int pos_col = searchers[ search ].pos_col;

	/* Stop if searcher finds another trail */
	int check;	
	check = atomicAdd(&accessMat( tainted, pos_row, pos_col ), 1);

	if ( check != 0 ) {
		search_flag = 1;
	}
	else {
		/* Annotate the trail */
		accessMat( trails, pos_row, pos_col ) = search;

		/* Compute the height */
		accessMat( heights, pos_row, pos_col ) = get_height( pos_row, pos_col, rows, columns, x_min, x_max, y_min, y_max );

		/* Locate the highest climbing direction */
		float local_max = accessMat( heights, pos_row, pos_col );
		int climbing_direction = 0;
		if ( pos_row > 0 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row-1, pos_col ) == INT_MIN ) 
				accessMat( heights, pos_row-1, pos_col ) = get_height( pos_row-1, pos_col, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row-1, pos_col ) > local_max ) {
				climbing_direction = 1;
				local_max = accessMat( heights, pos_row-1, pos_col );
			}
		}
		if ( pos_row < rows-1 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row+1, pos_col ) == INT_MIN )
				accessMat( heights, pos_row+1, pos_col ) = get_height( pos_row+1, pos_col, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row+1, pos_col ) > local_max ) {
				climbing_direction = 2;
				local_max = accessMat( heights, pos_row+1, pos_col );
			}
		}
		if ( pos_col > 0 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row, pos_col-1 ) == INT_MIN ) 
				accessMat( heights, pos_row, pos_col-1 ) = get_height( pos_row, pos_col-1, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row, pos_col-1 ) > local_max ) {
				climbing_direction = 3;
				local_max = accessMat( heights, pos_row, pos_col-1 );
			}
		}
		if ( pos_col < columns-1 ) {
			/* Compute the height in the neighbor if needed */
			if ( accessMat( heights, pos_row, pos_col+1 ) == INT_MIN ) 
				accessMat( heights, pos_row, pos_col+1 ) = get_height( pos_row, pos_col+1, rows, columns, x_min, x_max, y_min, y_max );

			/* Annotate the travelling direction if higher */
			if ( accessMat( heights, pos_row, pos_col+1 ) > local_max ) {
				climbing_direction = 4;
				local_max = accessMat( heights, pos_row, pos_col+1 );
			}
		}

		/* Stop if local maximum is reached */
		if ( climbing_direction == 0 ) {
			searchers[ search ].follows = search;
			search_flag = 1;
		}

		/* Move in the chosen direction: 0 does not change coordinates */
		switch( climbing_direction ) {
			case 1: pos_row--; break;
			case 2: pos_row++; break;
			case 3: pos_col--; break;
			case 4: pos_col++; break;
		}
		searchers[ search ].pos_row = pos_row;
		searchers[ search ].pos_col = pos_col;
	}

	/* Return a flag to indicate if search should stop */
	return search_flag;
}

#ifdef DEBUG
/* 
 * Function: Print the current state of the simulation 
 */
void print_heights( int rows, int columns, int *heights ) {
	/* 
	 * You don't need to optimize this function, it is only for pretty 
	 * printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;
	printf("Heights:\n");
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( heights, i, j ) != INT_MIN ) 
				printf(" %6d", accessMat( heights, i, j ) );
			else
				printf("       ");
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n\n");
}

void print_trails( int rows, int columns, int *trails ) {
	/* 
	 * You don't need to optimize this function, it is only for pretty 
	 * printing and debugging purposes.
	 * It is not compiled in the production versions of the program.
	 * Thus, it is never used when measuring times in the leaderboard
	 */
	int i,j;
	printf("Trails:\n");
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n");
	for( i=0; i<rows; i++ ) {
		printf("|");
		for( j=0; j<columns; j++ ) {
			char symbol;
			if ( accessMat( trails, i, j ) != -1 ) 
				printf("%7d", accessMat( trails, i, j ) );
			else
				printf("       ", accessMat( trails, i, j ) );
		}
		printf("|\n");
	}
	printf("+");
	for( j=0; j<columns; j++ ) printf("-------");
	printf("+\n\n");
}
#endif // DEBUG

/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<rows> <columns> <x_min> <x_max> <y_min> <y_max> <searchers_density> <short_rnd1> <short_rnd2> <short_rnd3>\n");
	fprintf(stderr,"\n");
}

__global__ void reductionMax(int* array, int size, int *result)
{
	// Compute the global position of the thread in the grid
	int globalPos = threadIdx.x + blockIdx.x * blockDim.x;

	// Shared memory: One element per thread in the block
	// Call this kernel with the proper third launching parameter
	extern __shared__ int buffer[ ];

	// Load array values in the shared memory (0 if out of the array)
	if ( globalPos < size ) { 
		buffer[ threadIdx.x ] = array[ globalPos ];
	}
	else buffer[ threadIdx.x ] = 0;

	// Wait for all the threads of the block to finish
	__syncthreads();

	// Reduction tree
	for( int step=blockDim.x/2; step>=1; step /= 2 ) {
		if ( threadIdx.x < step )
			if ( buffer[ threadIdx.x ] < buffer[ threadIdx.x + step ] )
				buffer[ threadIdx.x ] = buffer[ threadIdx.x + step ];
		__syncthreads();
	}

	// The maximum value of this block is on the first position of buffer
	if ( threadIdx.x == 0 )
		atomicMax( result, buffer[0] );
}

/*Kernel de inicializacion de searchers*/
__global__ void kernel_searchers_ini(Searcher *searchers, int num_searchers, int *total_steps){

	int search = threadIdx.x + (blockIdx.x * blockDim.x);

	if(search < num_searchers){
		searchers[ search ].steps = 0;
		searchers[ search ].follows = -1;
		total_steps[ search ] = 0;
	}
}

/*Kernel de llamada a la funcion del dispositivo climbing step*/
__global__ void kernel_climbing(int num_searchers, int rows, int columns, Searcher *searchers, int *heights, int *trails, int *tainted, float x_min, float x_max, float y_min, float y_max){

	int search = threadIdx.x + (blockIdx.x * blockDim.x);
	int search_flag = 0;
	if(search < num_searchers){		
			while( ! search_flag ) {
				search_flag = climbing_step( rows, columns, searchers, search, heights, trails, tainted, x_min, x_max, y_min, y_max );
			}
	}
}


/*Kernel para calcular los searchers que se deben seguir si el camino ya esta recorrido*/
__global__ void kernel_leading_follower(Searcher *searchers, int num_searchers){

	int search = threadIdx.x + (blockIdx.x * blockDim.x);

	if(search < num_searchers){
		int search_flag = 0;
		int parent = search;
		int follows_to = searchers[ parent ].follows;
		while( ! search_flag ) {
			if ( follows_to == parent ) search_flag = 1;
			else {
				parent = follows_to;
				follows_to = searchers[ parent ].follows;
			}
		}
		searchers[ search ].follows = follows_to;	
	}
}

/*Kernel para calcular pasos total y guardarlos en su correspondiente matriz*/
__global__ void kernel_total_steps(Searcher *searchers, int num_searchers, int *total_steps){

	int search = threadIdx.x + (blockIdx.x * blockDim.x);

	if(search < num_searchers){
		int pos_max = searchers[ search ].follows;
		atomicAdd(&total_steps[ pos_max ], searchers[ search ].steps ); //Suma atomica para evitar condicion de carrera al tratarse de lectura-escritura en mismas posiciones
	}
}


/*Kernel para realizar la parte de lectura de trails que se ha extraido de la funcion climbing_step con el fin de evitar la condicion de carrera producida por la propia escritura tambien realizada */
__global__ void kernel_climbing_fuera(Searcher *searchers, int *trails, int num_searchers, int columns){
		int search = threadIdx.x + (blockIdx.x * blockDim.x);

		if(search < num_searchers){
			int pos_row = searchers[ search ].pos_row;
			int pos_col = searchers[ search ].pos_col;
			searchers[ search ].follows = accessMat( trails, pos_row, pos_col);
		}
}

/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	// This eliminates the buffer of stdout, forcing the messages to be printed immediately
	setbuf(stdout,NULL);

	int i,j;

	// Simulation data
	int rows, columns;		// Matrix sizes
	float x_min, x_max;		// Limits of the terrain x coordinates
	float y_min, y_max;		// Limits of the terrain y coordinates

	float searchers_density;	// Density of hill climbing searchers
	unsigned short random_seq[3];	// Status of the random sequence

	int *heights;			// Heights of the terrain points
	int *trails;			// Searchers trace and trails
	int *tainted;			// Position found in a search
	int num_searchers;		// Number of searchers
	Searcher *searchers;		// Searchers data
	int *total_steps;		// Annotate accumulated steps to local maximums

	/* 1. Read simulation arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc != 11) {
		fprintf(stderr, "-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	rows = atoi( argv[1] );
	columns = atoi( argv[2] );
	x_min = atof( argv[3] );
	x_max = atof( argv[4] );
	y_min = atof( argv[5] );
	y_max = atof( argv[6] );
	searchers_density = atof( argv[7] );

	/* 1.3. Read random sequences initializer */
	for( i=0; i<3; i++ ) {
		random_seq[i] = (unsigned short)atoi( argv[8+i] );
	}


#ifdef DEBUG
	/* 1.4. Print arguments */
	printf("Arguments, Rows: %d, Columns: %d\n", rows, columns);
	printf("Arguments, x_range: ( %d, %d ), y_range( %d, %d )\n", x_min, x_max, y_min, y_max );
	printf("Arguments, searchers_density: %f\n", searchers_density );
	printf("Arguments, Init Random Sequence: %hu,%hu,%hu\n", random_seq[0], random_seq[1], random_seq[2]);
	printf("\n");
#endif // DEBUG


	/* 2. Start global timer */
	CHECK_CUDA_CALL( hipSetDevice(0) );
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 *
 */

	/* 3. Initialization */
	/* 3.1. Memory allocation */
	num_searchers = (int)( rows * columns * searchers_density );
	
	int nThreads = 1024;
	int nBlocks = num_searchers/nThreads; 
	int resto = num_searchers % nThreads;
	size_t tamShared= nThreads*sizeof(int); //Tamaño shared memory usado en reducciones
	if(resto != 0){
		nBlocks++;
	}

	searchers = (Searcher *)malloc( sizeof(Searcher) * num_searchers ); 
	total_steps = (int *)malloc( sizeof(int) * num_searchers ); 
	if ( searchers == NULL || total_steps == NULL ) {
		fprintf(stderr,"-- Error allocating searchers structures for size: %d\n", num_searchers );
		exit( EXIT_FAILURE );
	}

	heights = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );
	trails = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );
	tainted = (int *)malloc( sizeof(int) * (size_t)rows * (size_t)columns );
	if ( heights == NULL || trails == NULL || tainted == NULL ) {
		fprintf(stderr,"-- Error allocating terrain structures for size: %d x %d \n", rows, columns );
		exit( EXIT_FAILURE );
	}

	
	int search;
	Searcher *d_Searchers;
	int *d_total_steps;
	int *d_heights;
	int *d_trails;
	int *d_tainted;
	hipError_t error;
	
	
	
	error = hipMalloc((void**)&d_Searchers, sizeof(Searcher)*num_searchers);
	if ( error != hipSuccess )
		printf("ErrCUDA Malloc: %s\n", hipGetErrorString( error ) );

	error = hipMalloc((void**)&d_total_steps, sizeof(int)*num_searchers);
	if ( error != hipSuccess )
		printf("ErrCUDA Malloc: %s\n", hipGetErrorString( error ) );

	
	error=hipMalloc((void**)&d_trails,  sizeof(int) * (size_t)rows * (size_t)columns);
	if ( error != hipSuccess )
		printf("ErrCUDA Malloc: %s\n", hipGetErrorString( error ) );
	
	error = hipMalloc((void**)&d_heights,  sizeof(int) * (size_t)rows * (size_t)columns);
	if ( error != hipSuccess )
		printf("ErrCUDA Malloc: %s\n", hipGetErrorString( error ) );

	error = hipMalloc((void**)&d_tainted,  sizeof(int) * (size_t)rows * (size_t)columns);
	if ( error != hipSuccess )
		printf("ErrCUDA Malloc: %s\n", hipGetErrorString( error ) );

	
	/* 3.2. Terrain initialization */
	for( i=0; i<rows; i++ ) {
		for( j=0; j<columns; j++ ) {
			accessMat( heights, i, j ) = INT_MIN;
			accessMat( trails, i, j ) = -1;
			accessMat( tainted, i, j ) = 0;
		}
	}

	/* 3.3. Searchers initialization */
	for( search = 0; search < num_searchers; search++ ) {
		searchers[ search ].pos_row = (int)( rows * erand48( random_seq ) );
		searchers[ search ].pos_col = (int)( columns * erand48( random_seq ) );
	}
	/*Copia matrices a memoria del dispositivo*/
	hipMemcpy(d_tainted, tainted, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyHostToDevice);
	hipMemcpy(d_trails, trails, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyHostToDevice);
	hipMemcpy(d_heights, heights, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyHostToDevice);
	hipMemcpy(d_Searchers, searchers, sizeof(Searcher) * num_searchers, hipMemcpyHostToDevice);
	hipMemcpy(d_total_steps, total_steps, sizeof(int) * num_searchers, hipMemcpyHostToDevice);
	

	/*Llamada a kernel para inicializacion de searchers*/
	kernel_searchers_ini<<<nBlocks, nThreads>>>(d_Searchers, num_searchers, d_total_steps);
	error = hipGetLastError();
	if ( error != hipSuccess )
		printf("ErrCUDA Kernel Searchers: %s\n", hipGetErrorString( error ) );
	
	/* 4. Compute searchers climbing trails */	
	kernel_climbing<<<nBlocks, nThreads>>>(num_searchers, rows, columns, d_Searchers, d_heights, d_trails, d_tainted, x_min, x_max, y_min, y_max);
	error = hipGetLastError();
	if ( error != hipSuccess )
		printf("ErrCUDA Kernel Climbing: %s\n", hipGetErrorString( error ) );

	/*Kernel para realizar parte extraida de climbing_steps para evitar condicion de carrera*/
	kernel_climbing_fuera<<<nBlocks, nThreads>>>(d_Searchers, d_trails, num_searchers, columns);
	error = hipGetLastError();	
	if ( error != hipSuccess )
		printf("ErrCUDA Kernel Climb_Fuera: %s\n", hipGetErrorString( error ) );

	/* 5. Compute the leading follower of each searcher */
	kernel_leading_follower<<<nBlocks, nThreads>>>(d_Searchers, num_searchers);
	error = hipGetLastError();	
	if ( error != hipSuccess )
		printf("ErrCUDA Kernerl Leading: %s\n", hipGetErrorString( error ) );
	
	/* 6. Compute accumulated trail steps to each maximum */
	kernel_total_steps<<<nBlocks, nThreads>>>(d_Searchers, num_searchers, d_total_steps);
	error = hipGetLastError();	
	if ( error != hipSuccess )
		printf("ErrCUDA Kerner Total Steps: %s\n", hipGetErrorString( error ) );

	/*Copia matrices de memoria de dispositivo a memoria del host*/
	hipMemcpy(searchers, d_Searchers, sizeof(Searcher) * num_searchers, hipMemcpyDeviceToHost);
	hipMemcpy(tainted, d_tainted, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);
	hipMemcpy(heights, d_heights, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);
	hipMemcpy(trails, d_trails, sizeof(int) * (size_t)rows * (size_t)columns, hipMemcpyDeviceToHost);
	

	int max_accum_steps;
	int *dev_result;
	error = hipMalloc((void**)&dev_result, sizeof(int)*nBlocks);
	if ( error != hipSuccess )
		printf("ErrCUDA Malloc: %s\n", hipGetErrorString( error ) );

	/* Kernel para obtener max_accum_steps por reduccion */
	reductionMax<<<nBlocks,nThreads,tamShared>>>(d_total_steps,num_searchers, dev_result);
	error = hipGetLastError();	
	if ( error != hipSuccess )
		printf("ErrCUDA Kernel Reduction: %s\n", hipGetErrorString( error ) );
	hipMemcpy(&max_accum_steps, dev_result, sizeof(int), hipMemcpyDeviceToHost);

	/*Copia matrices de memoria de dispositivo a memoria del host*/
	hipMemcpy(total_steps, d_total_steps, sizeof(int) * num_searchers, hipMemcpyDeviceToHost);

	/*Liberacion memoria dinamica del dispositivo*/
	hipFree(d_Searchers);
	hipFree(d_total_steps);
	hipFree(d_trails);
	hipFree(d_heights);
	hipFree(d_tainted);
	hipFree(dev_result);
	
	/* 7. Compute statistical data */
	int num_local_max = 0;
	int max_height = INT_MIN;
	//int max_accum_steps = INT_MIN;
	int total_tainted = 0;
	unsigned long long int total_heights = 0;

	for( search = 0; search < num_searchers; search++ ) {
		/* If this searcher found a maximum, check the maximum value */
		if ( searchers[ search ].follows == search ) {
			num_local_max++;
			int pos_row = searchers[ search ].pos_row;
			int pos_col = searchers[ search ].pos_col;
			if ( max_height < accessMat( heights, pos_row, pos_col ) ) 
				max_height = accessMat( heights, pos_row, pos_col );
		}
	}
	

	for( i=0; i<rows; i++ ) {
		for( j=0; j<columns; j++ ) {
			if ( accessMat( tainted, i, j ) ) 
				total_tainted++;
		}
	}
	for( i=0; i<rows; i++ ) {
		for( j=0; j<columns; j++ ) {
			if ( accessMat( heights, i, j ) != INT_MIN ) 
				total_heights += accessMat( heights, i, j );
		}
	}
	
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 5. Stop global time */
	CHECK_CUDA_CALL( hipDeviceSynchronize() );
	ttotal = cp_Wtime() - ttotal;

	/* 6. Output for leaderboard */
	printf("\n");
	/* 6.1. Total computation time */
	printf("Time: %lf\n", ttotal );

	/* 6.2. Results: Statistics */
	printf("Result: %d, %d, %d, %d, %llu\n\n", 
			num_local_max,
			max_height,
			max_accum_steps,
			total_tainted,
			total_heights );
		
	/* 7. Free resources */	
	free( searchers );
	free( total_steps );
	free( heights );
	free( trails );
	free( tainted );

	/* 8. End */
	return 0;
}
